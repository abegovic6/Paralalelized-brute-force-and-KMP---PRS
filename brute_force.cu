#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "timerc.h"

#define SIZE 1024*1024*16
#define gerror(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void brute_force(char *text, char *pattern, int *match, int pattern_size, int text_size){
        int pid = threadIdx.x + blockIdx.x*blockDim.x;

        if (pid <= text_size - pattern_size){
            int flag = 1; 
            for (int i = 0; i < pattern_size; i++){
                if (text[pid+i] != pattern[i]){
                        flag = 0;
			break;
                }
            }
            match[pid] = flag;
        }
}


int cap_division(int x, int y){
    return (x + y - 1) / y;
}

void witness_array_cpu(char *pattern, int *witness_array, int pattern_size){
    if (pattern_size >2){
        witness_array[0] = 0;
        for (int i = 1; i<cap_division(pattern_size, 2); i++){
            for (int j=0; j<cap_division(pattern_size, 2); j++){
                if (pattern[j] != pattern[i+j]){
                    witness_array[i] = j;
                    break;
                }
            }
        }
    }else{
        witness_array[0] = 0;
    }
}

void failure_function_cpu(char *pattern, int *failure_function, int pattern_size){
    
    failure_function[0] = 0;
    
    int k = 1;
    int j = 0;
    
    while ( k < pattern_size){
        if (pattern[k] == pattern[j]){
            j ++;
            failure_function[k] = j;
            k ++;
        }else{
            if (j !=0){
                k = failure_function[k-1];
            }else{
                failure_function[k] =0;
                k++;
            }
        }
    }
}

void serial_string_matching_KMP(char *text, char *pattern, int pattern_size, int text_size, int *failure_function){
    int i = 0;
    int j = 0;
    
    while (i < text_size){
        if (pattern[j] == text[i]){
            j++;
            i++;
        }
        
        if (j == pattern_size){
            //printf("found at index %d \n", i-j);
            j = failure_function[j-1];
        }
        else if ( i < text_size && pattern[j] != text[i]){
            if (j != 0){
                j = failure_function[j-1];
            }else{
                i+=1;
            }
        }
        
    }
}




int main(){
    FILE *fp;
    FILE *fp2;
    char ch;
    fp = fopen("test.txt", "r");
    fp2 = fopen("pattern.txt", "r");
    
    char * text = (char *) malloc (SIZE*sizeof(char)); //size text buffer for text
    char * pattern = (char *) malloc (SIZE*sizeof(char));
    
    int * match; //size text buffeer for match array
    int size = 0;
    int pattern_size = 0;
    //int blocksize = 32;
    
    //intialized time
    float cpuTime;
    float gpuTime0;
    float gpuTime1;
    float gpuTime2;
    float gpuTime3;
    float cpuTime1;
    
    //read text to buffer
    while ((ch = getc(fp)) != EOF){
        text[size] = ch; 
        //match[size] = 0;
        size ++;
        if (size>=SIZE) break;
    }
    
    while ((ch =getc(fp2))!=EOF){
        pattern[pattern_size] = ch;
        pattern_size++;
    }
    
    size --;
    pattern_size--;
    printf("size %d \n", size);
    printf("pattern size %d \n", pattern_size);
    
    int *output = (int *) malloc (sizeof(int)*size);
    
    
    /*initialized match array*/
    match = (int *) malloc (size*sizeof(int));
    for (int i = 0; i < size; i++){
        match[i] = -1;
    }
    
    
    /*malloc wintess array*/
    int *witness_array = (int *)malloc(sizeof(int)*cap_division(pattern_size, 2));
    witness_array_cpu(pattern, witness_array, pattern_size);
    
    cstart();
    int *failure_function = (int *)malloc(sizeof(int)*(pattern_size));
    failure_function_cpu(pattern, failure_function, pattern_size);
    
    cend(&cpuTime);
    
    
    cstart();
    serial_string_matching_KMP(text, pattern, pattern_size, size, failure_function);
    cend(&cpuTime1);
    
    printf("CPU prepare time: %f", cpuTime);
    printf("KMP time: %f", cpuTime1);
    /* GPU init*/
    //text buffer in device
    char *dev_text;
    //pattern buffer in device
    char *dev_pattern;
    // match buffer in device
    int *dev_match;
    //output buffer in device
    int *dev_output;
    //witness array
    int *dev_witness;
    
    int number_of_blocks = 1
    if (size/pattern_size < 1024)
	number_of_blocks = (size/pattern_size + 1)/1024;
    
    gstart();
    hipMalloc((void **)&dev_text, size*sizeof(char));
    hipMalloc((void **)&dev_pattern, pattern_size*sizeof(char));
    hipMalloc((void **)&dev_match, size*sizeof(int));
    //hipMalloc((void **)&dev_output, sizeof(int)*size);
    hipMalloc((void **)&dev_witness, sizeof(int)*cap_division(pattern_size, 2));

    hipMemcpy(dev_text, text, size*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_pattern, pattern, pattern_size*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(dev_witness, witness_array, cap_division(pattern_size, 2)*sizeof(int), hipMemcpyHostToDevice);
    
    gend(&gpuTime0);
    
    gstart();
    
    brute_force<<<number_of_blocks, 1024>>>(dev_text, dev_pattern, dev_match, pattern_size, size);
    gend(&gpuTime2);
    
    gstart();
    hipMemcpy(match, dev_match, size*sizeof(int), hipMemcpyDeviceToHost);
    
    gend(&gpuTime3);
    
    if (flag ==1){
        printf("success");
    }else{
        printf ("error");
    }
    
    printf("\n");
    
    printf("<<<<output>>>> \n");
    for (int i = 0; i< number_of_blocks; i++){
        printf("%d ", blockoutput[i]);
    }
    printf("\n");
    */
    gerror( hipPeekAtLastError() );
    hipDeviceSynchronize();
    /*free memory*/
    hipFree(dev_text);
    hipFree(dev_pattern);
    hipFree(dev_match);
    hipFree(dev_output);
    hipFree(dev_witness);
    
    free(text);
    free(pattern);
    free(match);
    free(witness_array);
    free(failure_function);
    
    printf("CPUTIME: %f, GPUTIME0: %f, GPUTIME1: %f, GPUTIME2:%f, GPUTIME3:%f, TOTAL: %f", cpuTime,gpuTime0, gpuTime1, gpuTime2, gpuTime3, cpuTime+gpuTime1+gpuTime2 + gpuTime0+gpuTime3);
    
    //printf("CPUTIME: %f, GPUTIME0: %f, GPUTIME1: %f, GPUTIME3:%f, TOTAL: %f", cpuTime,gpuTime0, gpuTime1, gpuTime3, cpuTime+gpuTime1+gpuTime0+gpuTime3);
        
}